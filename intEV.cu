#include "hip/hip_runtime.h"
//=========================================
// Kernel code for computing Wilson loops on the device
//=========================================
#include <builtin_types.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "intT.h"
#include <math.h>


extern "C" void EVMean(double *EV, float4 *Wsscal_h, float4 *Wsferm_h, int n, int *WL, double T, int fermion);
#if profile == fixflux
 #define THREADS_PER_BLOCK 128
 #define MY_KERNEL_MAX_THREADS THREADS_PER_BLOCK
 #define MY_KERNEL_MIN_BLOCKS 4
#else
 #define THREADS_PER_BLOCK 256
 #if __CUDA_ARCH__ >= 200
  #define MY_KERNEL_MAX_THREADS (2 * THREADS_PER_BLOCK)
  #define MY_KERNEL_MIN_BLOCKS 3
 #else
  #define MY_KERNEL_MAX_THREADS (2 * THREADS_PER_BLOCK)
  #define MY_KERNEL_MIN_BLOCKS 2
 #endif
#endif



extern "C"
__device__ float expint(const float x)
//Evaluates the exponential integral Ei(x)=-E1(-x) assuming x<-1.
//This algorithm is an abbreviated version of Numerical Recipes expint().
//See Chapter 6 on Special Functions. We assume (x > 1.0) and include only 
//the relevant code. This bit of code is Lentz's algorithm (section 5.2 of NR).
{
  const int MAXIT = 400;
  const float EPS = 1.0e-6;
  const float BIG = 1.0e10;
  int i;
  float a, b, c, d, del, h, ans;
  b = -x + 1.0f;
  c = BIG;
  d = 1.0f/b;
  h = d;
  for (i = 1; i <= MAXIT; i++)
  {
	a = -(float)i*(float)i;
	b += 2.0f;
	d = 1.0f/(a*d+b); //Denominators cannot be zero
	c = b + a/c;
	del = c*d;
	h *= del;
	if (fabsf(del-1.0f) <= EPS)
	{
		ans = -h*__expf(x);
		return ans;
	}
  }
  return 0.0f;

}


extern "C"
__host__ __device__ float interp(float rho2, float *rho2sp, float4 *coefs)
//interpolation function for periodic spline profile
{
  int j;
  int upperi = Nspline-1, loweri=0;
  float rho2diff = 0.0f;
  float flambda;

  #ifndef __CUDA_ARCH__
        #warning __CUDA_ARCH__ Undefined!
	//printf("Printf error\n");
  #else
	#warning __CUDA_ARCH__ defined!
  #endif


    //Discover which interval to look in using a binary search
  if(rho2 < rho2sp[Nspline-1] && rho2 > rho2sp[0])
//  if(0)
  {
   	while(upperi-loweri > 1)
	{
		if(rho2 >= rho2sp[(upperi+loweri)/2]) loweri=(upperi+loweri)/2;
		else upperi = (upperi+loweri)/2;
  	}
  	//interpolate using the jth interval
  	j = loweri;
	rho2diff = rho2-rho2sp[j];
	//rho2diff=0.0;
  	flambda = coefs[j].x+rho2diff*(coefs[j].y+rho2diff*(coefs[j].z+rho2diff*coefs[j].w));
  }
  else
  {
	flambda=0.0f;
  }
  
  //*flambda= coefs[j].x+rho2diff*(coefs[j].y+rho2diff*(coefs[j].z+rho2diff*coefs[j].w));
  //*fprime = coefs[j].y+rho2diff*(2.0f*coefs[j].z + rho2diff*(3.0f*coefs[j].w));
  //*flambda=1.0f-exp(-1.0f*rho2);
  //*fprime=1.0f*exp(-1.0f*rho2);
  return flambda;

}



extern "C"
__host__ double EV (double T, void * p, int* WLlist) {
//Function for calling the Kernel, then computing the 
//Expectation value from the results of each worldline
  struct Wparams params = *(struct Wparams *) p;
  double EV;
  const int groupsize = 128;
  double rtT = sqrt((double)T);
  hipError_t errorcode;
  // call to integrate the function func
  if( verbosity >= 5)
  	printf("call to CUDA device\n");
  ExpectValue<<<params.nBlocks, params.nThreads>>>(params.Wsscal_d, params.Wsferm_d,
	params.worldlines,params.xcm, (float)params.F, (float)params.l2, (float)rtT, params.Nl, 
	params.Nppl, params.flcoefs, params.rho2sp, params.fermion);
  errorcode = hipGetLastError();
  if ( errorcode>0) printf("cuda getLastError EV(): %s\n", hipGetErrorString(errorcode));
  if (verbosity >= 6)
  	printf("return from CUDA\n");
  //Copy device memory back to host
  errorcode = hipMemcpy(params.Wsscal_h, params.Wsscal_d,
	params.Nl*sizeof(params.Wsscal_h[0]), hipMemcpyDeviceToHost);
  if(errorcode > 0) printf("cuda memcpy scal Error EV(): %s\n", hipGetErrorString(errorcode));
  if(params.fermion == 1)
  {
  	errorcode = hipMemcpy(params.Wsferm_h, params.Wsferm_d,
		params.Nl*sizeof(params.Wsferm_h[0]), hipMemcpyDeviceToHost);
	if(errorcode > 0) printf("cuda memcpy ferm Error EV(): %s\n",hipGetErrorString(errorcode));
  }
  //Compute the expectation value from the Wilson Loop data
  EV = 0.0;
  EVMean(&EV, params.Wsscal_h, params.Wsferm_h, groupsize,WLlist, T, params.fermion);
  //printf("EV=%f\n",EV);
  //result=exp(-m2*T)/(T*T*T)*(EV-1.0+1.0/6.0*TB*TB);
  //*SEout=(double) exp(-m2*T)/(T*T*T)*SE;
  //printf("%f %f \n",T,result);
  return EV;
}


extern "C"
__device__ float bump(const float x)
//Device version of the bump function
{
  //the 0.999 makes no numerical difference compared to 1.0
  //but seems to prevent some unpredictable, unspecified CUDA errors
  if(x*x < 0.999f) return __expf( -1.0f/(1.0f-x*x) );
  else return 0.0f;
}

extern "C"
__device__ float phi(const float x)
//Computes the \Phi function which is defined in the thesis
{
  const float onemx2 = 1.0f-x*x;
  //the 0.999 makes no numerical difference compared to 1.0
  //but seems to prevent some unpredictable, unspecified CUDA errors
  if( x < 0.999f )
  {
	return 1.0f - ( 0.5f/q2 )*( onemx2*__expf(-1.0f/onemx2) + expint(-1.0f/onemx2) );
  }
  else
	return 1.0f;
}

extern "C"
__device__ float chi(const float x, const float n, const float lambda)
//Computes the \Chi function which is defined in the thesis
{
  float ans;
  const float onemx2 = 1.0f-x*x;
  const float x2 = x*x;
  const float x4 = x2*x2;
  if(x <= -0.999f) ans = 0.0f;
  else if(x2 < 0.999f)
  {
	ans = 0.5f*(-onemx2*__expf(-1.0f/onemx2) - expint(-1.0f/onemx2));
	ans += 2.0f*n*tubedist/lambda*
		0.444f/( 1.0f + __expf(-(3.31f*x +
			5.25f*x*x2*sin(x)*cos( -0.907f*x2 - 1.29f*x4*x4 ))/cos(x)) );
	//an older, less precise approximation of the integral
	//ans += 2.0f*n*tubedist/lambda*
	//	(0.218f+0.393f*x*coshf(0.806f*x4-0.696f*x2+0.0902f)/coshf(0.825f*x2-0.0234f*x+0.375f));
  }
  else //for x >= 1.0f
  	ans = 2.0f*n*tubedist*q1/lambda;
  return ans;
}


extern "C"
__device__ float flperi(const float rho2, const float lambda2)
//f_lambda for periodic flux tube profile
{
  float expr, exprp1, exprap1, rho, lambda, flraf, f;
  int N = 2;
  int i;
  rho = sqrtf(rho2);
  lambda = sqrtf(lambda2);

  flraf = floorf(rho/tubedist);
  f = 0.0f;
  if((int)flraf-N > 0)
  {
	f=0.5f*(flraf-(float)N-1.0f)*(flraf-(float)N);
  }
  for(i = (int)flraf-N; i <= (int)flraf+N; i++)
  {
	if(i == 0)
	{
		expr = __expf(-rho/lambda);
		exprp1 = 1.0f+expr;
		f = -rho/tubedist*(expr/exprp1) - lambda/tubedist*log(exprp1);
	}
        else if(i > 0)
	{
		exprap1 = 1.0f+__expf(-(rho-(float)i*tubedist)/lambda);
		f = f+(float)i/exprap1;
	}
  }
  return tubedist/(lambda*log(2.0f))*f + 1.0f;
}



extern "C"
__device__ float ffixflux(const float rho2, const float lambda2)
//f_lambda for the fixflux profile
{
  const float lam = sqrtf(lambda2);
  const float lmlmin = (lam-lmin)/(tubedist-lmin);
  const float aml = (tubedist-lam)/(tubedist-lmin);
  const float n = floorf((sqrt(rho2)+tubedist/2.0f)/tubedist);
  float ans;
  if(rho2 <= tubedist*tubedist/4.0f)
  {
	ans = (1.0f-0.75f*lmlmin)*phi(2.0f*sqrt(rho2/lambda2)) + 3.0f*rho2/(tubedist*tubedist)*lmlmin;

  }
  else
  {
	ans = 1.0f+0.75f*(4.0f*rho2/(tubedist*tubedist)-1.0f)*lmlmin + 3.0f*n*(n-1.0f)*aml; 
	ans += 3.0f*lam/(q1*tubedist)*aml*chi(2.0f*(sqrt(rho2)-n*tubedist)/lam, n, lam);
	//ans = 1.01f;
  }
  return ans;
}


extern "C"
__device__ float flambda(float rho2, float lambda2, float4 *flcoefs, float *rho2sp)
//flambda(rho^2,lambda^2) defines the magnetic vector potential in cylindrical coordinates
{
  float f;

  switch(profile){
	case step: 
	  if(rho2 < lambda2)
		f = rho2/lambda2;
	  else
		f = 1.0f;
	  break;
	case smooth:
  	  f = rho2/(lambda2+rho2);
	  break;
	case quadratic:
	  if(rho2 < lambda2)
	  	f = rho2/lambda2*(2.0f-rho2/lambda2);
	  else
		f = 1.0f;
	  break;
	case gaussian:
	  f = 1.0f-exp(-rho2/lambda2);
	  break;
	case periodic:
	  f = flperi(rho2, lambda2);
	  break;
	case spline:
	  f = interp(rho2, rho2sp, flcoefs);
	  break;
	case fixflux:
	  f = ffixflux(rho2, lambda2);
	  break;
  }
  return f;
  //return rho2/lambda2;
} 


extern "C"
__device__ float fplperi(float rho2, float lambda2)
//f'_lambda for periodic profile
{
  float expr,exprp1,rho,lambda,flraf,f;
  float expral, expralp1;
  int N = 2;
  int i;
  rho = sqrtf(rho2);
  lambda = sqrtf(lambda2); 
  flraf = floorf(rho/tubedist);
  f=0.0f;
  for(i = (int)flraf-N; i <= (int)flraf+N; i++)
  {
	if(i == 0)
	{
		expr = __expf(-rho/lambda);
		exprp1 = 1.0f+expr;
		f = expr/(tubedist*exprp1*exprp1);
	}
	else if(i > 0)
	{
		expral = __expf(-(rho-(float)i*tubedist)/lambda);
		expralp1 = 1.0f+expral;
		f = f +((float)i/rho)*expral/(expralp1*expralp1);
	}
  }
  return tubedist/(2.0f*lambda2*log(2.0f))*f;
}




extern "C"
__device__ float fpfixflux(const float rho2, const float lambda2)
//f'_lambda(rho^2) for the fixflux field profile
{
  const float lam = sqrtf(lambda2);
  const float lmlmin = (lam-lmin)/(tubedist-lmin);
  const float rho = sqrtf(rho2);
  const float n = floorf((rho+tubedist/2.0f)/tubedist);
  const float a2 = tubedist*tubedist;
  const float aml = (tubedist - lam)/(tubedist - lmin);
  float ans;
  if(rho <= tubedist/2.0f)
  {
	ans = 2.0f/(lambda2*q2)*(1.0f-0.75f*lmlmin)*bump(2.0f*rho/lam);
	ans += 3.0f/a2*lmlmin;
  }
  else
  {
	ans = 3.0f/a2*lmlmin + 6.0f/(q1*lam*tubedist)*aml*bump(2.0f*(rho-n*tubedist)/lam);
	//ans = 3.0f/(tubedist*tubedist)*lmlmin + 6.0f/(q1*lam*tubedist)*
	//	(tubedist-lam)/(tubedist-lmin)*bump(2.0f*(rho-n*tubedist)/lam);
  }
  //ans = 1.0f;
  return ans;
}


extern "C"
__device__ float fplambda(const float rho2, const float lambda2, float4 *flcoefs, float *rho2sp)
//fplambda(rho^2,lambda^2) defines the magnetic vector potential derivative 
// wrt rho^2 in cylindrical coordinates
{
  float f, fjunk;
  f=1.0f;
  switch(profile){
	case step: 
	  if(rho2 < lambda2)
		f = 1.0f/lambda2;
	  else
		f = 0.0f;
	  break;
	case smooth:
  	  f = lambda2/((lambda2+rho2)*(lambda2+rho2));
	  break;
	case quadratic:
	  if(rho2<lambda2)
	  	f = 2.0f/lambda2*(1.0f-rho2/lambda2);
	  else
		f = 0.0f;
	  break;
	case gaussian:
	  f = 1.0f/lambda2*exp(-1.0f*rho2/lambda2);
	  break;
	case periodic:
	  f = fplperi(rho2, lambda2);
	  break;
	case spline:
	  fjunk = __sinf(sqrtf(rho2)*pi);
	  if(fjunk>100.0f)
		f = 0.0f;
	  else
	  {
		f = __expf(-1.0f*fjunk*fjunk/lambda2)/lambda2;
		//f=1.0f/lambda2*exp(-1.0f*rho2/lambda2);
	  }
	  break;
	case fixflux:
	  f = fpfixflux(rho2, lambda2);
	  break;
  }
  return f;
  //return 1.0f/lambda2;
  //return lambda2/((lambda2+rho2)*(lambda2+rho2));
} 

extern "C"
__device__ void Idt(float *scalI, float *fermI, float4 Ai, 
	const float l2, float4 *flcoefs, float *rho2sp, int fermion)
//Computes the integral over t from 0 to 1 in the scalar and fermion Wilson loop factors
{
	int i;
	const int n = 50;        //number of points in point-to-point proper time integral
	float t,  rhoi2; //proper time and rho squared
	const float h = 1.0f/((float) n);  //distance between points in integral
	float4 xiscal, xiferm;     //scalar and fermi integrands
	if (Ai.x<1.0e-8) Ai.x = 1.0e-8;
	if (Ai.y<1.0e-8) Ai.y = 1.0e-8;
	if (Ai.z<1.0e-8) Ai.z = 1.0e-8;
	float Aip1 = Ai.x+2.0f*Ai.y+Ai.z;  //rho^2 for the final point
	if(Aip1<1.0e-8) Aip1 = 1.0e-8;
	//if(profile == periodic && Aip1 > 10.0f*tubedist) Aip1 = 1.0e-8;
	//Begin the Simpson's method algorithm
	xiscal.x = flambda(Ai.x,l2,flcoefs,rho2sp)/Ai.x + flambda(Aip1,l2,flcoefs,rho2sp)/Aip1;
	xiscal.y = 0.0f;
	xiscal.z = 0.0f;
	if (fermion == 1)
	{
	  xiferm.x = fplambda(Ai.x, l2, flcoefs, rho2sp) + fplambda(Aip1, l2, flcoefs, rho2sp);
	  xiferm.y = 0.0f;
	  xiferm.z = 0.0f;
	}
	for(i = 1; i < n; i++)
	{
		t = (float)i*h;
		//rho2 at the point
		rhoi2 = Ai.x + 2.0f*Ai.y*t + Ai.z*t*t;
		if(rhoi2 < 1.0e-10) rhoi2 = 1.0e-10;
		//if(profile == periodic && rhoi2 > 10.0f*tubedist) rhoi2 = 1.0e-8;
		if(i%2==0) 
		{
			xiscal.z += flambda(rhoi2, l2, flcoefs, rho2sp)/rhoi2;
			if(fermion == 1)
			  xiferm.z += fplambda(rhoi2, l2, flcoefs, rho2sp);
		}
		else 
		{
			xiscal.y += flambda(rhoi2, l2, flcoefs, rho2sp)/rhoi2;
			if(fermion == 1)
			  xiferm.y += fplambda(rhoi2, l2, flcoefs, rho2sp);
		}
	
	}
	*scalI = (xiscal.x + 2.0f*xiscal.z + 4.0f*xiscal.y)*h/3.0f;
	if(fermion == 1)
	  *fermI = (xiferm.x + 2.0f*xiferm.z + 4.0f*xiferm.y)*h/3.0f;
	//*fermI=1.0f/l2;
}

extern "C"
__device__ void getzp1(float4 *zip1, float4 *worldlines, 
	float rtT, float4 xcm, int i, int inx, int Nppl)
//Function for determining the next point on the 
//worldline loop for each of the sub loops
{
  int inxp1;
  //get the next worldline index for the N/2 group
  if(i%2 == 1){
	if(i == Nppl-1)
	{
		inxp1 = inx*Nppl+1;
	}
	else
	{
		inxp1 = inx*Nppl+i+2;
	}
  }
  //get the next worldline index for the first N/4 group
  else if(i%4 == 0){
	if(i == Nppl-4)
	{
		inxp1 = inx*Nppl;
	}
	else
	{
		inxp1 = inx*Nppl+i+4;
	}
  }
  //get the next worldline index for the second N/4 group
  else if((i-2)%2 == 0){
	if(i == Nppl-2)
	{
		inxp1 = inx*Nppl+2;
	}
	else
	{
		inxp1 = inx*Nppl+i+4;
	}
  }
  //compute the next point
  zip1->x = xcm.x + rtT*worldlines[inxp1].x;
  zip1->y = xcm.y + rtT*worldlines[inxp1].y;
  zip1->z = xcm.z + rtT*worldlines[inxp1].z;

}

extern "C"
__device__ void WilsonLoop(float4 *worldlines, float4 *Wsscal, float4 *Wsferm, 
	float4 xcm, int inx, float F, float l2, float rtT, 
	int Nppl, float4 *flcoefs, float *rho2sp, int fermion)
//Returns the Wilson loop value
{
	int i;
	//const float e = 1.0;
        float4 WLstemp, WLftemp;
	float4 zi, zip1;
	float4 Ai;
	float xyyx;
	float scalI, fermI;
	//Compute the scalar contribution
	WLstemp.x = 0.0f; WLstemp.y = 0.0f; WLstemp.z = 0.0f;
	WLftemp.x = 0.0f; WLftemp.y = 0.0f; WLftemp.z = 0.0f;
	for(i = 0; i < Nppl; i++){
		//Compute the scaled, shifted coordinate
		zi.x = xcm.x + rtT*worldlines[inx*Nppl+i].x;
		zi.y = xcm.y + rtT*worldlines[inx*Nppl+i].y;
		getzp1(&zip1, worldlines, rtT, xcm, i, inx, Nppl);
		//Ai Bi and Ci coefficients for the rho2 polynomial
		Ai.x = zi.x*zi.x + zi.y*zi.y;
		Ai.y = zi.x*(zip1.x-zi.x)+zi.y*(zip1.y-zi.y);
		Ai.z = (zip1.x-zi.x)*(zip1.x-zi.x) 
			+ (zip1.y-zi.y)*(zip1.y-zi.y);
		Idt(&scalI, &fermI, Ai, l2, flcoefs, rho2sp, fermion);
		//scalI=1.0f/l2;
		//Compute the contribution to the N/2 integral
		xyyx = (zi.x*zip1.y-zi.y*zip1.x);
		if(i%2 == 1){
			WLstemp.x += xyyx*scalI;
			WLftemp.x += fermI;
		}
		//Compute the contribution to the first N/4 integral
		else if(i%4 == 0){
			WLstemp.z += xyyx*scalI;
			WLftemp.z += fermI;
		}
		//Compute the contribution to the second N/4 integral
		else if((i-2)%2 == 0){
			WLstemp.y += xyyx*scalI;
			WLftemp.y += fermI;
		}
	}
	Wsscal[inx].x = F*WLstemp.x;
	Wsscal[inx].y = F*WLstemp.y;
	Wsscal[inx].z = F*WLstemp.z;
	if( fermion == 1)
	{
	  Wsferm[inx].x = 2.0f*F*WLftemp.x*rtT*rtT/(Nppl/2.0f);
	  Wsferm[inx].y = 2.0f*F*WLftemp.y*rtT*rtT/(Nppl/4.0f);
	  Wsferm[inx].z = 2.0f*F*WLftemp.z*rtT*rtT/(Nppl/4.0f);
	}
	//Wsferm[inx].x=2.0f*F/l2*rtT*rtT;
	//Wsferm[inx].y=2.0f*F/l2*rtT*rtT;
	//Wsferm[inx].z=2.0f*F/l2*rtT*rtT;

	//Wsferm[inx].x=1.0f;
	//Wsferm[inx].y=1.0f;
	//Wsferm[inx].z=1.0f;

	
}

__global__ void 
__launch_bounds__(MY_KERNEL_MAX_THREADS, MY_KERNEL_MIN_BLOCKS)
ExpectValue(float4 *Wsscal, float4 *Wsferm, float4 *worldlines, 
	float4 xcm, float F, float l2, float rtT, int Nl, int Nppl, float4 *flcoefs, float *rho2sp, int fermion)
//Each thread computes the Wilson loop value for a single 
//worldline
{
        int inx = blockIdx.x * blockDim.x + threadIdx.x;       
        WilsonLoop(worldlines, Wsscal, Wsferm, xcm, inx, F, l2, rtT, Nppl, flcoefs, rho2sp, fermion);     
}




